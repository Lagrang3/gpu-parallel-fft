#include "hip/hip_runtime.h"
#define BOOST_TEST_MODULE 1D - FFTW - Tests

#include <boost/math/special_functions/fpclassify.hpp>
#include <boost/test/unit_test.hpp>
#include <cmath>
#include <complex>
#include <gpfft/detail/cufft_wrapper.hpp>
#include <gpfft/detail/fftw_wrapper.hpp>
#include <vector>

using cd = std::complex<double>;
using gpfft::FFT_type;
namespace math = boost::math;

BOOST_AUTO_TEST_CASE(small_transforms_fftw)
{
    const double pi = acos(-1.0);
    std::vector<cd> out;

    out = gpfft::FFTW3<FFT_type::forward>({3});
    BOOST_CHECK(out.size() == 1);
    BOOST_CHECK_SMALL(std::abs(out[0] - cd{3, 0}), 1e-12);
    for (auto c : out)
        BOOST_CHECK(math::isnan(abs(c)) == false);

    out = gpfft::FFTW3<FFT_type::backward>({3});
    BOOST_CHECK(out.size() == 1);
    BOOST_CHECK_SMALL(std::abs(out[0] - cd{3, 0}), 1e-12);
    for (auto c : out)
        BOOST_CHECK(math::isnan(abs(c)) == false);

    out = gpfft::FFTW3<FFT_type::forward>({1, 1});
    BOOST_CHECK(out.size() == 2);
    BOOST_CHECK_SMALL(std::abs(out[0] - cd{2, 0}), 1e-12);
    BOOST_CHECK_SMALL(std::abs(out[1] - cd{0, 0}), 1e-12);
    for (auto c : out)
        BOOST_CHECK(math::isnan(abs(c)) == false);

    {
        out = gpfft::FFTW3<FFT_type::forward>({1, 1, 1});
        cd w{cos(2 * pi / 3), sin(2 * pi / 3)}, w2 = w * w;
        BOOST_CHECK(out.size() == 3);
        BOOST_CHECK_SMALL(std::abs(out[0] - cd{3, 0}), 1e-12);
        BOOST_CHECK_SMALL(std::abs(out[1] - 1. - w - w2), 1e-12);
        BOOST_CHECK_SMALL(std::abs(out[2] - 1. - w2 - w2 * w2), 1e-12);
        for (auto c : out)
            BOOST_CHECK(math::isnan(abs(c)) == false);
    }
}
BOOST_AUTO_TEST_CASE(small_transforms_cufft)
{
    const double pi = acos(-1.0);
    std::vector<cd> out;

    out = gpfft::cuFFT<FFT_type::forward>({3});
    BOOST_CHECK(out.size() == 1);
    BOOST_CHECK_SMALL(std::abs(out[0] - cd{3, 0}), 1e-12);
    for (auto c : out)
        BOOST_CHECK(math::isnan(abs(c)) == false);

    out = gpfft::cuFFT<FFT_type::backward>({3});
    BOOST_CHECK(out.size() == 1);
    BOOST_CHECK_SMALL(std::abs(out[0] - cd{3, 0}), 1e-12);
    for (auto c : out)
        BOOST_CHECK(math::isnan(abs(c)) == false);

    out = gpfft::cuFFT<FFT_type::forward>({1, 1});
    BOOST_CHECK(out.size() == 2);
    BOOST_CHECK_SMALL(std::abs(out[0] - cd{2, 0}), 1e-12);
    BOOST_CHECK_SMALL(std::abs(out[1] - cd{0, 0}), 1e-12);
    for (auto c : out)
        BOOST_CHECK(math::isnan(abs(c)) == false);

    {
        out = gpfft::cuFFT<FFT_type::forward>({1, 1, 1});
        cd w{cos(2 * pi / 3), sin(2 * pi / 3)}, w2 = w * w;
        BOOST_CHECK(out.size() == 3);
        BOOST_CHECK_SMALL(std::abs(out[0] - cd{3, 0}), 1e-12);
        BOOST_CHECK_SMALL(std::abs(out[1] - 1. - w - w2), 1e-12);
        BOOST_CHECK_SMALL(std::abs(out[2] - 1. - w2 - w2 * w2), 1e-12);
        for (auto c : out)
            BOOST_CHECK(math::isnan(abs(c)) == false);
    }
}
