#include "hip/hip_runtime.h"
#define BOOST_TEST_MODULE 3D_FFTW_Tests

#include <boost/math/special_functions/fpclassify.hpp>
#include <boost/mpi.hpp>
#include <boost/test/unit_test.hpp>
#include <gpfft/fft_type.hpp>
#include <gpfft/parallel_buffer.hpp>

#include <cmath>
#include <complex>
#include <random>
#include <vector>

#include <fftw3-mpi.h>

using cd = std::complex<double>;
using gpfft::FFT_type;

namespace ut = boost::unit_test;
namespace mpi = boost::mpi;
namespace math = boost::math;

struct fixture
{
    fixture() { fftw_mpi_init(); }

    ~fixture() {}

    static mpi::environment env;
    static mpi::communicator world;
    static std::default_random_engine rng;
};

mpi::environment fixture::env;
mpi::communicator fixture::world;
std::default_random_engine fixture::rng{111};

BOOST_TEST_GLOBAL_FIXTURE(fixture);

BOOST_AUTO_TEST_CASE(gpfft_stability)
{
    BOOST_REQUIRE(fixture::world.size() == 2);
    const int nc = 14;
    const int localn = nc / fixture::world.size();

    BOOST_REQUIRE(nc == localn * fixture::world.size());

    gpfft::parallel_buff_3D<cd> A(fixture::world, {nc, nc, nc});
    std::uniform_real_distribution<double> U(0, 1);

    for (size_t i = 0; i < A.size(); ++i)
        A[i] = U(fixture::rng);

    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.local_FFT<FFT_type::forward>();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.transpose_yz();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.local_FFT<FFT_type::forward>();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.transpose_yz();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.transpose_xz();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.local_FFT<FFT_type::forward>();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);

    A.transpose_xz();
    for (size_t i = 0; i < A.size(); ++i)
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);
}

BOOST_AUTO_TEST_CASE(gpfft_vs_fftw3_parallel)
{
    BOOST_REQUIRE(fixture::world.size() == 2);
    const int nc = 14;
    gpfft::parallel_buff_3D<cd> A(fixture::world, {nc, nc, nc});
    std::uniform_real_distribution<double> U(0, 1);

    for (size_t i = 0; i < A.size(); ++i)
        A[i] = U(fixture::rng);

    // fftw3-3d

    ptrdiff_t local_n, local_start;
    ptrdiff_t alloc_local = fftw_mpi_local_size_3d(nc, nc, nc, fixture::world,
                                                   &local_n, &local_start);
    fftw_complex* data = fftw_alloc_complex(alloc_local);

    fftw_plan p = fftw_mpi_plan_dft_3d(nc, nc, nc, data, data, fixture::world,
                                       FFTW_FORWARD, FFTW_ESTIMATE);

    const size_t local_size = local_n * nc * nc;
    BOOST_REQUIRE(local_size == A.size());

    for (size_t i = 0; i < local_size; ++i)
    {
        data[i][0] = A[i].real(), data[i][1] = A[i].imag();
    }

    fftw_execute(p);

    A.FFT3D<gpfft::FFT_type::forward>();
    for (size_t i = 0; i < A.size(); ++i)
    {
        BOOST_REQUIRE(math::isnan(std::abs(A[i])) == false);
    }

    double diff = 0;
    for (size_t i = 0; i < local_size; ++i)
    {
        diff += std::abs(A[i] - cd(data[i][0], data[i][1]));
    }

    mpi::all_reduce(fixture::world, diff, std::plus<double>());

    fftw_free(data);
    fftw_destroy_plan(p);

    BOOST_CHECK(math::isnan(diff) == false);
    BOOST_CHECK_SMALL(diff, 1e-11);
}
