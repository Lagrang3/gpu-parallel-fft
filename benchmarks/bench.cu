#include "hip/hip_runtime.h"
#include <boost/mpi.hpp>
#include <gpfft/gpfft.hpp>
#include <boost/math/special_functions/fpclassify.hpp>

#include <cassert>
#include <complex>
#include <random>
#include <vector>
#include <chrono>

#include <fftw3-mpi.h>

using cd = std::complex<double>;
using gpfft::FFT_type;
using microseconds = std::chrono::microseconds;

namespace math = boost::math;
namespace mpi = boost::mpi;

int main()
{
    // initialize
    
    mpi::environment env;
    mpi::communicator world;
    std::default_random_engine rng{111};
    
    const int Nx= 512;
    assert(Nx % world.size() == 0);
    gpfft::parallel_buff_3D<cd> A(world, {Nx,Nx,Nx});
    
    std::uniform_real_distribution<double> U(0, 1);

    for (size_t i = 0; i < A.size(); ++i)
        A[i] = U(rng);
    
    // .. code here for fftw
    ptrdiff_t local_n, local_start;
    ptrdiff_t alloc_local = fftw_mpi_local_size_3d(Nx,Nx,Nx,world,
                                                   &local_n, &local_start);
    fftw_complex* data = fftw_alloc_complex(alloc_local);

    fftw_plan p = fftw_mpi_plan_dft_3d(Nx,Nx,Nx, data, data,world,
                                       FFTW_FORWARD, FFTW_ESTIMATE);

    const size_t local_size = local_n * Nx*Nx;
    assert(local_size == A.size());

    for (size_t i = 0; i < local_size; ++i)
    {
        data[i][0] = A[i].real(), data[i][1] = A[i].imag();
    }

    auto t1 = std::chrono::high_resolution_clock::now();
    fftw_execute(p);
    auto dt = std::chrono::high_resolution_clock::now() - t1;
    double musec_fftw = std::chrono::duration_cast<microseconds>(dt).count();
    
    t1 = std::chrono::high_resolution_clock::now();
    A.FFT3D<gpfft::FFT_type::forward>();
    dt = std::chrono::high_resolution_clock::now() - t1;
    double musec_cufft = std::chrono::duration_cast<microseconds>(dt).count();
    
    // compare outputs
    for (size_t i = 0; i < A.size(); ++i)
    {
        assert(math::isnan(std::abs(A[i])) == false);
    }
    double diff = 0;
    for (size_t i = 0; i < local_size; ++i)
    {
        diff += std::abs(A[i] - cd(data[i][0], data[i][1]));
    }
    mpi::all_reduce(world, diff, std::plus<double>());
    assert(math::isnan(diff) == false);
    
    
    // release resources
    fftw_free(data);
    fftw_destroy_plan(p);
    
    if(world.rank()==0)
    std::cout 
        << "Time (fftw): " << musec_fftw*1e-3 << " ms\n"
        << "Time (cufft): " << musec_cufft*1e-3 << " ms\n"
        << "Diff: " << diff << '\n';
    
    return 0;
}
